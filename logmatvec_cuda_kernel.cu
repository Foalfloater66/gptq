#include <hip/hip_runtime.h>
#include <stdint.h> // For int64_t
#include <algorithm> // For std::max, std::min

// Clamping helper (optional, if exponents aren't pre-clamped)
// __device__ inline int clamp_exp(int exp, int min_exp, int max_exp) {
//     return max(min_exp, min(exp, max_exp));
// }

// Core Bitshift Operation
// x: quantized activation (int)
// y: weight exponent (int) - Handles positive (left shift) and negative (right shift)
// z: weight sign (signed char: +1, -1, 0)
// accumulator: pointer to the 64-bit accumulator
__device__ inline void accumulate_bitshift(const int x, const int y, const signed char z, int64_t* accumulator) {
    if (z == 0 || x == 0) { // Handle zero weight or activation
        return;
    }

    int64_t shifted_val;
    // Use standard C++ casting for clarity
    int64_t x_64 = static_cast<int64_t>(x);

    if (y >= 0) {
        // Left shift for positive exponents
        // Basic overflow check: if y is too large, result is likely 0 or incorrect anyway
        // A more robust check might compare y against (63 - position of highest set bit in x)
        if (y < 64) { // Avoid shifting by 64 or more bits
           shifted_val = (x_64 << y);
        } else {
           shifted_val = 0; // Or handle as error/saturation
        }
    } else {
        // Right shift for negative exponents
        // Shifting by negative amount is UB in C++, use positive amount
        int shift_amount = -y;
        if (shift_amount < 64) { // Avoid shifting by 64 or more bits
            shifted_val = (x_64 >> shift_amount);
        } else {
            // If shifting by 64 or more, result depends on sign of x
            shifted_val = (x_64 < 0) ? -1 : 0;
        }
    }

    // Apply sign using conditional expression
    *accumulator += (z > 0) ? shifted_val : -shifted_val;
}


// Helper function to unpack two 4-bit codes (sign+exponent) from an int8 byte
__device__ inline void unpack_4bit_codes(const int8_t packed_byte, uint8_t& code1, uint8_t& code2) {
    // Cast to unsigned char to avoid sign extension issues with right shift
    uint8_t unsigned_byte = static_cast<uint8_t>(packed_byte);
    code1 = (unsigned_byte >> 4) & 0x0F; // High nibble (first weight code)
    code2 = unsigned_byte & 0x0F;        // Low nibble (second weight code)
}

// Kernel for Log-Quantized Matrix (W) x Linear-Quantized Vector (a)
// Uses bundled packed 4-bit codes (1 sign + 3 exponent)
__global__ void LogMatVecKernelPacked4bit(
    const int* __restrict__ a_quant,          // Quantized activations [InFeatures]
    const int8_t* __restrict__ w_packed_4bit, // Packed 4-bit codes [OutFeatures * InFeatures/2]
    float* __restrict__ output,               // Output vector [OutFeatures]
    const float delta_lsb,                   // Activation scaling factor
    const int min_exp,                       // Minimum exponent value for unmapping (maps to exp_map=0)
    const int in_features,
    const int out_features
) {
    // Each block computes one output feature
    const int output_row = blockIdx.x;

    if (output_row >= out_features) {
        return;
    }

    // Accumulator for the dot product (use 64-bit to prevent overflow)
    int64_t accumulator = 0;

    // Each thread sums a portion of the dot product, processing two weights at a time
    // The loop iterates over packed exponent bytes
    int packed_in_features = in_features / 2; // Number of packed bytes per row
    for (int packed_idx = threadIdx.x; packed_idx < packed_in_features; packed_idx += blockDim.x) {
        // Calculate index for packed exponent matrix (row-major)
        int packed_weight_idx = output_row * packed_in_features + packed_idx;
        // Calculate base index for signs and activations (corresponding to first weight in pair)
        int base_idx = packed_idx * 2;

        // Calculate index for packed weight matrix (row-major)
        // int packed_weight_idx = output_row * packed_in_features + packed_idx; // Duplicate removed
        // Calculate base index for activations (corresponding to first weight in pair)
        // int base_idx = packed_idx * 2; // Duplicate removed

        // Read packed byte and unpack 4-bit codes
        int8_t packed_byte = w_packed_4bit[packed_weight_idx];
        uint8_t code1, code2;
        unpack_4bit_codes(packed_byte, code1, code2);

        // Read corresponding activations
        int activation1 = a_quant[base_idx];
        int activation2 = a_quant[base_idx + 1];

        // --- Process first weight (code1) ---
        if (code1 != 0) { // Check for special zero code
            // Decode sign and exponent map
            signed char sign1 = (code1 & 0x08) ? -1 : 1; // MSB (bit 3) is sign (1=neg, 0=pos)
            uint8_t exp_map1 = code1 & 0x07; // 3 LSBs are exponent map
            // Unmap exponent (handle the offset for positive values)
            // Positive codes 1-7 map to exp_map 0-6 -> exponents min_exp to max_exp-1
            // Negative codes 8-15 map to exp_map 0-7 -> exponents min_exp to max_exp
            int exponent1;
            if (sign1 > 0) { // Positive (codes 1-7 map to exp_map 0-6)
                 exponent1 = static_cast<int>(exp_map1 - 1) + min_exp; // Map 1->min_exp, 7->max_exp-1
            } else { // Negative (codes 8-15 map to exp_map 0-7)
                 exponent1 = static_cast<int>(exp_map1) + min_exp; // Map 0->min_exp, 7->max_exp
            }
            accumulate_bitshift(activation1, exponent1, sign1, &accumulator);
        }

        // --- Process second weight (code2) ---
        if (code2 != 0) { // Check for special zero code
            // Decode sign and exponent map
            signed char sign2 = (code2 & 0x08) ? -1 : 1;
            uint8_t exp_map2 = code2 & 0x07;
            // Unmap exponent
            int exponent2;
             if (sign2 > 0) {
                 exponent2 = static_cast<int>(exp_map2 - 1) + min_exp;
            } else {
                 exponent2 = static_cast<int>(exp_map2) + min_exp;
            }
            accumulate_bitshift(activation2, exponent2, sign2, &accumulator);
        }
    }

    // --- Block-level reduction using shared memory (remains the same) ---
    // Allocate shared memory dynamically based on kernel launch parameter
    extern __shared__ int64_t sdata[];
    sdata[threadIdx.x] = accumulator;
    __syncthreads(); // Wait for all threads to write to shared memory

    // Reduce within the block (works for any blockDim.x)
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        // Only threads in the first half of the current range participate
        if (threadIdx.x < offset) {
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        __syncthreads(); // Synchronize after each reduction step
    }

    // Lead thread (threadIdx.x == 0) writes the final scaled result
    if (threadIdx.x == 0) {
        // Cast the final 64-bit integer result to float before scaling
        output[output_row] = static_cast<float>(sdata[0]) * delta_lsb;
    }
}


// --- Kernel Launcher ---
// This function is defined in the .cu file and called by the .cpp file.
// It sets up the <<<...>>> kernel launch syntax.
void LogMatVecKernelLauncher(
    const int* a_quant,
    const int8_t* w_packed_4bit, // Changed parameter name
    float* output,
    const float delta_lsb,
    const int min_exp,
    const int in_features,
    const int out_features,
    const dim3 blocks,
    const dim3 threads,
    const size_t shared_mem_size,
    // const int min_exp, // Remove from the end
    hipStream_t stream)
{
    // Launch the __global__ kernel for packed 4-bit codes
    LogMatVecKernelPacked4bit<<<blocks, threads, shared_mem_size, stream>>>(
        a_quant,
        w_packed_4bit, // Pass packed codes
        output,
        delta_lsb,
        min_exp,       // Pass min_exp for unmapping
        in_features,
        out_features
    );
}


// ============================================================================
// Kernel Version using Bundled 4-bit Codes but Float Multiplication
// ============================================================================

// Kernel for Log-Quantized Matrix (W) x Linear-Quantized Vector (a)
// Uses bundled packed 4-bit codes BUT performs float multiplication internally
__global__ void LogMatVecKernelBundled4bit_FloatMul( // New kernel name
    const int* __restrict__ a_quant,          // Quantized activations [InFeatures]
    const int8_t* __restrict__ w_packed_4bit, // Packed 4-bit codes [OutFeatures * InFeatures/2]
    float* __restrict__ output,               // Output vector [OutFeatures]
    const float delta_lsb,                   // Activation scaling factor
    const int min_exp,                       // Minimum exponent value for unmapping
    const int in_features,
    const int out_features
) {
    // Each block computes one output feature
    const int output_row = blockIdx.x;

    if (output_row >= out_features) {
        return;
    }

    // Accumulator for the dot product (use double for float accumulation precision)
    double accumulator = 0.0;

    // Each thread sums a portion of the dot product, processing two weights at a time
    int packed_in_features = in_features / 2;
    for (int packed_idx = threadIdx.x; packed_idx < packed_in_features; packed_idx += blockDim.x) {
        int packed_weight_idx = output_row * packed_in_features + packed_idx;
        int base_idx = packed_idx * 2;

        // Read packed byte and unpack 4-bit codes
        int8_t packed_byte = w_packed_4bit[packed_weight_idx];
        uint8_t code1, code2;
        unpack_4bit_codes(packed_byte, code1, code2); // Use the same unpacker

        // Read corresponding activations
        int activation1_int = a_quant[base_idx];
        int activation2_int = a_quant[base_idx + 1];

        // --- Process first weight (code1) using float multiplication ---
        if (code1 != 0) { // Check for special zero code
            // Decode sign and exponent map
            signed char sign1_char = (code1 & 0x08) ? -1 : 1;
            uint8_t exp_map1 = code1 & 0x07;
            // Unmap exponent
            int exponent1;
            if (sign1_char > 0) { exponent1 = static_cast<int>(exp_map1 - 1) + min_exp; }
            else { exponent1 = static_cast<int>(exp_map1) + min_exp; }

            // Calculate float weight value
            float weight1_float = powf(2.0f, static_cast<float>(exponent1));
            // Multiply float weight by activation (cast activation to double)
            double term1 = static_cast<double>(activation1_int) * static_cast<double>(weight1_float);
            accumulator += (sign1_char > 0) ? term1 : -term1;
        }

        // --- Process second weight (code2) using float multiplication ---
        if (code2 != 0) { // Check for special zero code
            // Decode sign and exponent map
            signed char sign2_char = (code2 & 0x08) ? -1 : 1;
            uint8_t exp_map2 = code2 & 0x07;
            // Unmap exponent
            int exponent2;
            if (sign2_char > 0) { exponent2 = static_cast<int>(exp_map2 - 1) + min_exp; }
            else { exponent2 = static_cast<int>(exp_map2) + min_exp; }

            // Calculate float weight value
            float weight2_float = powf(2.0f, static_cast<float>(exponent2));
            // Multiply float weight by activation
            double term2 = static_cast<double>(activation2_int) * static_cast<double>(weight2_float);
            accumulator += (sign2_char > 0) ? term2 : -term2;
        }
    }

    // --- Block-level reduction using shared memory (using double) ---
    extern __shared__ double sdata_double[]; // Use double for shared memory
    sdata_double[threadIdx.x] = accumulator;
    __syncthreads();

    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        if (threadIdx.x < offset) {
            sdata_double[threadIdx.x] += sdata_double[threadIdx.x + offset];
        }
        __syncthreads();
    }

    // Lead thread writes the final scaled result
    if (threadIdx.x == 0) {
        // Cast the final double result to float before scaling
        output[output_row] = static_cast<float>(sdata_double[0]) * delta_lsb;
    }
}


// --- Kernel Launcher for Bundled 4-bit Float Multiplication Version ---
void LogMatVecKernelLauncher_BundledFloatMul( // New launcher name
    const int* a_quant,
    const int8_t* w_packed_4bit,
    float* output,
    const float delta_lsb,
    const int min_exp,
    const int in_features,
    const int out_features,
    const dim3 blocks,
    const dim3 threads,
    const size_t shared_mem_size, // Base size, will be adjusted
    hipStream_t stream)
{
    // Adjust shared memory size for double accumulator
    size_t shared_mem_double_size = threads.x * sizeof(double);

    // Launch the __global__ kernel for bundled 4-bit with float multiplication
    LogMatVecKernelBundled4bit_FloatMul<<<blocks, threads, shared_mem_double_size, stream>>>( // Call new kernel
        a_quant,
        w_packed_4bit,
        output,
        delta_lsb,
        min_exp,
        in_features,
        out_features
    );
}
