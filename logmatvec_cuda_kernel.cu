#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdint.h> // For int64_t
#include <algorithm> // For std::max, std::min

// Clamping helper (optional, if exponents aren't pre-clamped)
// __device__ inline int clamp_exp(int exp, int min_exp, int max_exp) {
//     return max(min_exp, min(exp, max_exp));
// }

// Core Bitshift Operation
// x: quantized activation (int)
// y: weight exponent (int) - Handles positive (left shift) and negative (right shift)
// z: weight sign (signed char: +1, -1, 0)
// accumulator: pointer to the 64-bit accumulator
__device__ inline void accumulate_bitshift(const int x, const int y, const signed char z, int64_t* accumulator) {
    if (z == 0 || x == 0) { // Handle zero weight or activation
        return;
    }

    int64_t shifted_val;
    // Use standard C++ casting for clarity
    int64_t x_64 = static_cast<int64_t>(x);

    if (y >= 0) {
        // Left shift for positive exponents
        // Basic overflow check: if y is too large, result is likely 0 or incorrect anyway
        // A more robust check might compare y against (63 - position of highest set bit in x)
        if (y < 64) { // Avoid shifting by 64 or more bits
           shifted_val = (x_64 << y);
        } else {
           shifted_val = 0; // Or handle as error/saturation
        }
    } else {
        // Right shift for negative exponents
        // Shifting by negative amount is UB in C++, use positive amount
        int shift_amount = -y;
        if (shift_amount < 64) { // Avoid shifting by 64 or more bits
            shifted_val = (x_64 >> shift_amount);
        } else {
            // If shifting by 64 or more, result depends on sign of x
            shifted_val = (x_64 < 0) ? -1 : 0;
        }
    }

    // Apply sign using conditional expression
    *accumulator += (z > 0) ? shifted_val : -shifted_val;
}


// Helper function to unpack two 4-bit values from an int8 byte
__device__ inline void unpack_4bit(const int8_t packed_byte, uint8_t& val1, uint8_t& val2) {
    // Cast to unsigned char to avoid sign extension issues with right shift
    uint8_t unsigned_byte = static_cast<uint8_t>(packed_byte);
    val1 = (unsigned_byte >> 4) & 0x0F; // High nibble
    val2 = unsigned_byte & 0x0F;        // Low nibble
}

// Kernel for Log-Quantized Matrix (W) x Linear-Quantized Vector (a)
// Uses packed 4-bit exponents and separate 8-bit signs
__global__ void LogMatVecKernelPacked4bit(
    const int* __restrict__ a_quant,          // Quantized activations [InFeatures]
    const int8_t* __restrict__ w_packed_exp,  // Packed 4-bit mapped exponents [OutFeatures * InFeatures/2]
    const signed char* __restrict__ w_sign,   // Weight signs [OutFeatures * InFeatures]
    float* __restrict__ output,               // Output vector [OutFeatures]
    const float delta_lsb,                   // Activation scaling factor
    const int min_exp,                       // Minimum exponent value for unmapping
    const int in_features,
    const int out_features
) {
    // Each block computes one output feature
    const int output_row = blockIdx.x;

    if (output_row >= out_features) {
        return;
    }

    // Accumulator for the dot product (use 64-bit to prevent overflow)
    int64_t accumulator = 0;

    // Each thread sums a portion of the dot product, processing two weights at a time
    // The loop iterates over packed exponent bytes
    int packed_in_features = in_features / 2; // Number of packed bytes per row
    for (int packed_idx = threadIdx.x; packed_idx < packed_in_features; packed_idx += blockDim.x) {
        // Calculate index for packed exponent matrix (row-major)
        int packed_weight_idx = output_row * packed_in_features + packed_idx;
        // Calculate base index for signs and activations (corresponding to first weight in pair)
        int base_idx = packed_idx * 2;

        // Read packed byte and unpack mapped exponents
        int8_t packed_byte = w_packed_exp[packed_weight_idx];
        uint8_t mapped_exp1, mapped_exp2;
        unpack_4bit(packed_byte, mapped_exp1, mapped_exp2);

        // Read corresponding signs
        signed char sign1 = w_sign[output_row * in_features + base_idx];
        signed char sign2 = w_sign[output_row * in_features + base_idx + 1];

        // Read corresponding activations
        int activation1 = a_quant[base_idx];
        int activation2 = a_quant[base_idx + 1];

        // --- Process first weight in pair ---
        if (sign1 != 0) {
            // Unmap the 4-bit value back to the actual exponent
            int exponent1 = static_cast<int>(mapped_exp1) + min_exp;
            accumulate_bitshift(activation1, exponent1, sign1, &accumulator);
        }

        // --- Process second weight in pair ---
        if (sign2 != 0) {
            // Unmap the 4-bit value back to the actual exponent
            int exponent2 = static_cast<int>(mapped_exp2) + min_exp;
            accumulate_bitshift(activation2, exponent2, sign2, &accumulator);
        }
    }

    // --- Block-level reduction using shared memory (remains the same) ---
    // Allocate shared memory dynamically based on kernel launch parameter
    extern __shared__ int64_t sdata[];
    sdata[threadIdx.x] = accumulator;
    __syncthreads(); // Wait for all threads to write to shared memory

    // Reduce within the block (works for any blockDim.x)
    for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {
        // Only threads in the first half of the current range participate
        if (threadIdx.x < offset) {
            sdata[threadIdx.x] += sdata[threadIdx.x + offset];
        }
        __syncthreads(); // Synchronize after each reduction step
    }

    // Lead thread (threadIdx.x == 0) writes the final scaled result
    if (threadIdx.x == 0) {
        // Cast the final 64-bit integer result to float before scaling
        output[output_row] = static_cast<float>(sdata[0]) * delta_lsb;
    }
}


// --- Kernel Launcher ---
// This function is defined in the .cu file and called by the .cpp file.
// It sets up the <<<...>>> kernel launch syntax.
void LogMatVecKernelLauncher(
    const int* a_quant,
    const int* w_exp,
    const signed char* w_sign,
    float* output,
    const float delta_lsb,
    const int in_features,
    const int out_features,
    const dim3 blocks,
    const dim3 threads,
    const size_t shared_mem_size,
    const int min_exp, // Added min_exp
    hipStream_t stream)
{
    // Launch the __global__ kernel for packed 4-bit
    LogMatVecKernelPacked4bit<<<blocks, threads, shared_mem_size, stream>>>(
        a_quant,
        w_packed_exp, // Pass packed exponents
        w_sign,       // Pass signs
        output,
        delta_lsb,
        min_exp,      // Pass min_exp for unmapping
        in_features,
        out_features
    );
}
